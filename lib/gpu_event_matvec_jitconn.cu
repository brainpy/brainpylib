#include "hip/hip_runtime.h"
//
// Created by adadu on 2022/11/30.
//

#include "gpu_event_matvec_jitconn.cuh"

namespace brainpy_lib {
    namespace {

        template<typename T, const int BLOCK_SIZE>
        __global__ void _event_jitconn_prob_homo(
                const int *event_ids,  /* event */
                const int &event_num,
                const unsigned int conn_seed,  /* matrix */
                const float conn_prob,
                const T &weight,
                const unsigned int num_row,  /* shape */
                const unsigned int num_col,
                T *out  /* output */
        ) {
            __shared__ int shEvents[BLOCK_SIZE];

            const unsigned int idx = threadIdx.x;
            const unsigned int row_i = blockIdx.x * blockDim.x + idx;

            if (idx < event_num)
                shEvents[idx] = event_ids[idx];
            __syncthreads();

            T sum = 0;
            int event_read_i = 0;
            while (true) {
#pragma unroll
                for (int sh_i = 0; sh_i < BLOCK_SIZE; sh_i++) {
                    if (sh_i + event_read_i < event_num) {
                        if (lfsr113_double(conn_seed + num_col * row_i + shEvents[sh_i]) < conn_prob)
                            sum += weight;
                    }
                }
                event_read_i += BLOCK_SIZE;
                if (event_read_i < event_num) {
                    if (idx + event_read_i < event_num)
                        shEvents[idx] = event_ids[idx + event_read_i];
                    __syncthreads();
                } else {
                    break;
                }
            }
            if (row_i < num_row) {
                out[row_i] = sum;
            }
        }

        template<typename T, const int BLOCK_SIZE>
        __global__ void _event_jitconn_transpose_prob_homo(
                const int *event_ids,  /* event */
                const int &event_num,
                const unsigned int conn_seed,  /* matrix */
                const float conn_prob,
                const T &weight,
                const unsigned int num_row,  /* shape */
                const unsigned int num_col,
                T *out  /* output */
        ) {
            __shared__ int shEvents[BLOCK_SIZE];

            const unsigned int idx = threadIdx.x;
            const unsigned int col_i = blockIdx.x * blockDim.x + idx;

            if (idx < event_num)
                shEvents[idx] = event_ids[idx];
            __syncthreads();

            T sum = 0;
            int event_read_i = 0;
            while (true) {
#pragma unroll
                for (int sh_i = 0; sh_i < BLOCK_SIZE; sh_i++) {
                    if (sh_i + event_read_i < event_num) {
                        if (lfsr113_double(conn_seed + num_col * shEvents[sh_i] + col_i) < conn_prob)
                            sum += weight;
                    }
                }
                event_read_i += BLOCK_SIZE;
                if (event_read_i < event_num) {
                    if (event_read_i + idx < event_num)
                        shEvents[idx] = event_ids[event_read_i + idx];
                    __syncthreads();
                } else {
                    break;
                }
            }
            if (col_i < num_col) {
                out[col_i] = sum;
            }
        }

        template<typename T>
        inline void event_jitconn_prob_homo(hipStream_t stream,
                                            void **buffers,
                                            const char *opaque,
                                            std::size_t opaque_len) {
            // size
            const JITConnProbCHomoWDescriptor &d = *UnpackDescriptor<JITConnProbCHomoWDescriptor>(opaque, opaque_len);
            const unsigned int n_row = d.n_row;
            const unsigned int n_col = d.n_col;
            const unsigned int conn_seed = d.seed;
            const float conn_prob = d.prob;
            const bool transpose = d.transpose;

            // data
            const int *event_ids = reinterpret_cast<const int *>(buffers[0]);
            const int *event_num = reinterpret_cast<const int *>(buffers[1]);
            const T *weight = reinterpret_cast<const T *>(buffers[2]);
            T *y = reinterpret_cast<T *>(buffers[3]);

            // processing
            const int block_dim = 256;
            const int grid_dim = (n_col + block_dim - 1) / block_dim;
            if (transpose) {
                hipMemset(y, 0, sizeof(T) * n_col);
                _event_jitconn_transpose_prob_homo<T, block_dim><<<grid_dim, block_dim, 0, stream>>>(
                        event_ids, event_num[0], conn_seed, conn_prob, weight[0], n_row, n_col, y
                );
            } else {
                hipMemset(y, 0, sizeof(T) * n_row);
                _event_jitconn_prob_homo<T, block_dim><<<grid_dim, block_dim, 0, stream>>>(
                        event_ids, event_num[0], conn_seed, conn_prob, weight[0], n_row, n_col, y
                );
            }
            ThrowIfError(hipGetLastError());
        }


        template<typename T, const int BLOCK_SIZE>
        __global__ void _event_jitconn_prob_uniform(
                const int *event_ids,  /* event */
                const int &event_num,
                const unsigned int conn_seed,  /* matrix */
                const float conn_prob,
                const float w_min,
                const float w_range,
                const unsigned int num_row,  /* shape */
                const unsigned int num_col,
                T *out  /* output */
        ) {
            __shared__ int shEvents[BLOCK_SIZE];

            const unsigned int idx = threadIdx.x;
            const unsigned int row_i = blockIdx.x * blockDim.x + idx;

            if (idx < event_num)
                shEvents[idx] = event_ids[idx];
            __syncthreads();

            T sum = 0;
            int event_read_i = 0;
            while (true) {
#pragma unroll
                for (int sh_i = 0; sh_i < BLOCK_SIZE; sh_i++) {
                    if (sh_i + event_read_i < event_num) {
                        unsigned int seed = conn_seed + num_col * row_i + shEvents[sh_i];
                        if (lfsr113_double(seed) < conn_prob)
                            sum += (taus88_double(seed) * w_range + w_min);
                    }
                }
                event_read_i += BLOCK_SIZE;
                if (event_read_i < event_num) {
                    if (idx + event_read_i < event_num)
                        shEvents[idx] = event_ids[idx + event_read_i];
                    __syncthreads();
                } else {
                    break;
                }
            }
            if (row_i < num_row) {
                out[row_i] = sum;
            }
        }


        template<typename T, const int BLOCK_SIZE>
        __global__ void _event_jitconn_transpose_prob_uniform(
                const int *event_ids,  /* event */
                const int &event_num,
                const unsigned int conn_seed,  /* matrix */
                const float conn_prob,
                const float w_min,
                const float w_range,
                const unsigned int num_row,  /* shape */
                const unsigned int num_col,
                T *out  /* output */
        ) {
            __shared__ int shEvents[BLOCK_SIZE];

            const unsigned int idx = threadIdx.x;
            const unsigned int col_i = blockIdx.x * blockDim.x + idx;

            if (idx < event_num)
                shEvents[idx] = event_ids[idx];
            __syncthreads();

            T sum = 0;
            int event_read_i = 0;
            while (true) {
#pragma unroll
                for (int sh_i = 0; sh_i < BLOCK_SIZE; sh_i++) {
                    if (sh_i + event_read_i < event_num) {
                        unsigned int seed = conn_seed + num_col * shEvents[sh_i] + col_i;
                        if (lfsr113_double(seed) < conn_prob)
                            sum += (taus88_double(seed) * w_range + w_min);
                    }
                }
                event_read_i += BLOCK_SIZE;
                if (event_read_i < event_num) {
                    if (event_read_i + idx < event_num)
                        shEvents[idx] = event_ids[event_read_i + idx];
                    __syncthreads();
                } else {
                    break;
                }
            }
            if (col_i < num_col) {
                out[col_i] = sum;
            }
        }


        template<typename T>
        inline void event_jitconn_prob_uniform(hipStream_t stream,
                                               void **buffers,
                                               const char *opaque,
                                               std::size_t opaque_len) {
            // size
            const JITConnProbCUniformWDescriptor &d = *UnpackDescriptor<JITConnProbCUniformWDescriptor>(opaque,
                                                                                                        opaque_len);
            const unsigned int n_row = d.n_row;
            const unsigned int n_col = d.n_col;
            const unsigned int conn_seed = d.seed;
            const float conn_prob = d.prob;
            const float w_min = d.w_min;
            const float w_range = d.w_range;
            const bool transpose = d.transpose;

            // data
            const int *event_ids = reinterpret_cast<const int *>(buffers[0]);
            const int *event_num = reinterpret_cast<const int *>(buffers[1]);
            T *y = reinterpret_cast<T *>(buffers[2]);

            // processing
            const int block_dim = 256;
            const int grid_dim = (n_col + block_dim - 1) / block_dim;
            if (transpose) {
                hipMemset(y, 0, sizeof(T) * n_col);
                _event_jitconn_transpose_prob_uniform<T, block_dim><<<grid_dim, block_dim, 0, stream>>>(
                        event_ids, event_num[0], conn_seed, conn_prob, w_min, w_range, n_row, n_col, y
                );
            } else {
                hipMemset(y, 0, sizeof(T) * n_row);
                _event_jitconn_transpose_prob_uniform<T, block_dim><<<grid_dim, block_dim, 0, stream>>>(
                        event_ids, event_num[0], conn_seed, conn_prob, w_min, w_range, n_row, n_col, y
                );
            }
            ThrowIfError(hipGetLastError());
        }

        template<typename T, const int BLOCK_SIZE>
        __global__ void _event_jitconn_prob_normal(
                const int *event_ids,  /* event */
                const int &event_num,
                const unsigned int conn_seed,  /* matrix */
                const float conn_prob,
                const float w_mu,
                const float w_sigma,
                const unsigned int num_row,  /* shape */
                const unsigned int num_col,
                T *out  /* output */
        ) {
            __shared__ int shEvents[BLOCK_SIZE];

            const unsigned int idx = threadIdx.x;
            const unsigned int row_i = blockIdx.x * blockDim.x + idx;

            if (idx < event_num)
                shEvents[idx] = event_ids[idx];
            __syncthreads();

            T sum = 0;
            int event_read_i = 0;
            while (true) {
#pragma unroll
                for (int sh_i = 0; sh_i < BLOCK_SIZE; sh_i++) {
                    if (sh_i + event_read_i < event_num) {
                        unsigned int seed = conn_seed + num_col * row_i + shEvents[sh_i];
                        if (lfsr113_double(seed) < conn_prob) {
                            double u = taus88_double(seed);
                            double v = taus88_double((seed * seed) >> 5);
                            double z = sqrt(-2 * log(u)) * cos(2 * M_PI * v);
                            sum += (w_mu + z * w_sigma);
                        }
                    }
                }
                event_read_i += BLOCK_SIZE;
                if (event_read_i < event_num) {
                    if (idx + event_read_i < event_num)
                        shEvents[idx] = event_ids[idx + event_read_i];
                    __syncthreads();
                } else {
                    break;
                }
            }
            if (row_i < num_row) {
                out[row_i] = sum;
            }
        }

        template<typename T, const int BLOCK_SIZE>
        __global__ void _event_jitconn_transpose_prob_normal(
                const int *event_ids,  /* event */
                const int &event_num,
                const unsigned int conn_seed,  /* matrix */
                const float conn_prob,
                const float w_mu,
                const float w_sigma,
                const unsigned int num_row,  /* shape */
                const unsigned int num_col,
                T *out  /* output */
        ) {
            __shared__ int shEvents[BLOCK_SIZE];

            const unsigned int idx = threadIdx.x;
            const unsigned int col_i = blockIdx.x * blockDim.x + idx;

            if (idx < event_num)
                shEvents[idx] = event_ids[idx];
            __syncthreads();

            T sum = 0;
            int event_read_i = 0;
            while (true) {
#pragma unroll
                for (int sh_i = 0; sh_i < BLOCK_SIZE; sh_i++) {
                    if (sh_i + event_read_i < event_num) {
                        unsigned int seed = conn_seed + num_col * shEvents[sh_i] + col_i;
                        if (lfsr113_double(seed) < conn_prob) {
                            double u = taus88_double(seed);
                            double v = taus88_double((seed * seed) >> 5);
                            double z = sqrt(-2 * log(u)) * cos(2 * M_PI * v);
                            sum += (w_mu + z * w_sigma);
                        }
                    }
                }
                event_read_i += BLOCK_SIZE;
                if (event_read_i < event_num) {
                    if (event_read_i + idx < event_num)
                        shEvents[idx] = event_ids[event_read_i + idx];
                    __syncthreads();
                } else {
                    break;
                }
            }
            if (col_i < num_col) {
                out[col_i] = sum;
            }
        }


        template<typename T>
        inline void event_jitconn_prob_normal(hipStream_t stream,
                                              void **buffers,
                                              const char *opaque,
                                              std::size_t opaque_len) {
            // size
            const JITConnProbCNormalWDescriptor &d = *UnpackDescriptor<JITConnProbCNormalWDescriptor>(opaque,
                                                                                                      opaque_len);
            const unsigned int n_row = d.n_row;
            const unsigned int n_col = d.n_col;
            const unsigned int conn_seed = d.seed;
            const float conn_prob = d.prob;
            const float w_mu = d.w_mu;
            const float w_sigma = d.w_sigma;
            const bool transpose = d.transpose;

            // data
            const int *event_ids = reinterpret_cast<const int *>(buffers[0]);
            const int *event_num = reinterpret_cast<const int *>(buffers[1]);
            T *y = reinterpret_cast<T *>(buffers[2]);

            // processing
            const int block_dim = 256;
            const int grid_dim = (n_col + block_dim - 1) / block_dim;
            if (transpose) {
                hipMemset(y, 0, sizeof(T) * n_col);
                _event_jitconn_transpose_prob_normal<T, block_dim><<<grid_dim, block_dim, 0, stream>>>(
                        event_ids, event_num[0], conn_seed, conn_prob, w_mu, w_sigma, n_row, n_col, y
                );
            } else {
                hipMemset(y, 0, sizeof(T) * n_row);
                _event_jitconn_transpose_prob_normal<T, block_dim><<<grid_dim, block_dim, 0, stream>>>(
                        event_ids, event_num[0], conn_seed, conn_prob, w_mu, w_sigma, n_row, n_col, y
                );
            }
            ThrowIfError(hipGetLastError());
        }


    }

    void event_matvec_jitconn_prob_homo_float(hipStream_t stream, void **buffers,
                                              const char *opaque, std::size_t opaque_len) {
        event_jitconn_prob_homo<float>(stream, buffers, opaque, opaque_len);
    }

    void event_matvec_jitconn_prob_homo_double(hipStream_t stream, void **buffers,
                                               const char *opaque, std::size_t opaque_len) {
        event_jitconn_prob_homo<double>(stream, buffers, opaque, opaque_len);
    }

    void event_matvec_jitconn_prob_uniform_float(hipStream_t stream, void **buffers,
                                                 const char *opaque, std::size_t opaque_len) {
        event_jitconn_prob_uniform<float>(stream, buffers, opaque, opaque_len);
    }

    void event_matvec_jitconn_prob_uniform_double(hipStream_t stream, void **buffers,
                                                  const char *opaque, std::size_t opaque_len) {
        event_jitconn_prob_uniform<double>(stream, buffers, opaque, opaque_len);
    }


    void event_matvec_jitconn_prob_normal_float(hipStream_t stream, void **buffers,
                                                const char *opaque, std::size_t opaque_len) {
        event_jitconn_prob_normal<float>(stream, buffers, opaque, opaque_len);
    }

    void event_matvec_jitconn_prob_normal_double(hipStream_t stream, void **buffers,
                                                 const char *opaque, std::size_t opaque_len) {
        event_jitconn_prob_normal<double>(stream, buffers, opaque, opaque_len);
    }


}

