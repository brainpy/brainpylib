#include "hip/hip_runtime.h"
//
// Created by adadu on 2022/11/30.
//

#include "gpu_event_matvec_jitconn.cuh"

namespace brainpy_lib {
    namespace {

        template<typename T, const int BLOCK_SIZE>
        __global__ void _event_mv_C_fixedprob_W_homo(
                const int *event_ids,  /* event */
                const int &event_num,
                const unsigned int conn_seed,  /* matrix */
                const float conn_prob,
                const T &weight,
                const unsigned int num_row,  /* shape */
                const unsigned int num_col,
                T *out  /* output */
        ) {
            __shared__ int shEvents[BLOCK_SIZE];

            const unsigned int idx = threadIdx.x;
            const unsigned int row_i = blockIdx.x * blockDim.x + idx;

            if (idx < event_num)
                shEvents[idx] = event_ids[idx];
            __syncthreads();

            T sum = 0;
            int event_read_i = 0;
            while (true) {
#pragma unroll
                for (int sh_i = 0; sh_i < BLOCK_SIZE; sh_i++) {
                    if (sh_i + event_read_i < event_num) {
                        if (lfsr113_double(conn_seed + num_col * row_i + shEvents[sh_i]) < conn_prob)
                            sum += weight;
                    }
                }
                event_read_i += BLOCK_SIZE;
                if (event_read_i < event_num) {
                    if (idx + event_read_i < event_num)
                        shEvents[idx] = event_ids[idx + event_read_i];
                    __syncthreads();
                } else {
                    break;
                }
            }
            if (row_i < num_row) {
                out[row_i] = sum;
            }
        }

        template<typename T, const int BLOCK_SIZE>
        __global__ void _event_mv_transpose_C_fixedprob_W_homo(
                const int *event_ids,  /* event */
                const int &event_num,
                const unsigned int conn_seed,  /* matrix */
                const float conn_prob,
                const T &weight,
                const unsigned int num_row,  /* shape */
                const unsigned int num_col,
                T *out  /* output */
        ) {
            __shared__ int shEvents[BLOCK_SIZE];

            const unsigned int idx = threadIdx.x;
            const unsigned int col_i = blockIdx.x * blockDim.x + idx;

            if (idx < event_num)
                shEvents[idx] = event_ids[idx];
            __syncthreads();

            T sum = 0;
            int event_read_i = 0;
            while (true) {
#pragma unroll
                for (int sh_i = 0; sh_i < BLOCK_SIZE; sh_i++) {
                    if (sh_i + event_read_i < event_num) {
                        if (lfsr113_double(conn_seed + num_col * shEvents[sh_i] + col_i) < conn_prob)
                            sum += weight;
                    }
                }
                event_read_i += BLOCK_SIZE;
                if (event_read_i < event_num) {
                    if (event_read_i + idx < event_num)
                        shEvents[idx] = event_ids[event_read_i + idx];
                    __syncthreads();
                } else {
                    break;
                }
            }
            if (col_i < num_col) {
                out[col_i] = sum;
            }
        }

//        <typename T>
//        __global__ void _event_mv_transpose_C_fixedprob_W_uniform(
//                /* event */
//                const int *event_ids,
//                const int &event_num,
//
//                /* matrix */
//                const unsigned int weight_seed,
//                const unsigned int conn_seed,
//                const float conn_prob,
//
//                /* shape */
//                const unsigned int num_row,
//                const unsigned int num_col,
//
//                /* output */
//                T *out
//        ) {
//
//        }
//
//        <typename T>
//        __global__ void _event_mv_transpose_C_fixedprob_W_normal(
//                /* event */
//                const int *event_ids,
//                const int &event_num,
//
//                /* matrix */
//                const unsigned int weight_seed,
//                const unsigned int conn_seed,
//                const float conn_prob,
//
//                /* shape */
//                const unsigned int num_row,
//                const unsigned int num_col,
//
//                /* output */
//                T *out
//        ) {
//
//        }


        template<typename T>
        inline void event_mv_C_fixedprob_W_homo(hipStream_t stream,
                                                void **buffers,
                                                const char *opaque,
                                                std::size_t opaque_len) {
            // size
            const EventMVRandomDescriptor &d = *UnpackDescriptor<EventMVRandomDescriptor>(opaque, opaque_len);
            const unsigned int n_row = d.n_row;
            const unsigned int n_col = d.n_col;
            const unsigned int conn_seed = d.seed;
            const float conn_prob = d.prob;
            const bool transpose = d.transpose;

            // data
            const int *event_ids = reinterpret_cast<const int *>(buffers[0]);
            const int *event_num = reinterpret_cast<const int *>(buffers[1]);
            const T *weight = reinterpret_cast<const T *>(buffers[2]);
            T *y = reinterpret_cast<T *>(buffers[3]);

            // processing
            const int block_dim = 256;
            const int grid_dim = (n_col + block_dim - 1) / block_dim;
            if (transpose) {
                hipMemset(y, 0, sizeof(T) * n_col);
                _event_mv_transpose_C_fixedprob_W_homo<T, block_dim><<<grid_dim, block_dim, 0, stream>>>(
                        event_ids, event_num[0], conn_seed, conn_prob, weight[0], n_row, n_col, y
                );
            } else {
                hipMemset(y, 0, sizeof(T) * n_row);
                _event_mv_C_fixedprob_W_homo<T, block_dim><<<grid_dim, block_dim, 0, stream>>>(
                        event_ids, event_num[0], conn_seed, conn_prob, weight[0], n_row, n_col, y
                );
            }
            ThrowIfError(hipGetLastError());
        }

    }

    void event_mv_C_fixedprob_W_homo_float(hipStream_t stream, void **buffers,
                                           const char *opaque, std::size_t opaque_len) {
        event_mv_C_fixedprob_W_homo<float>(stream, buffers, opaque, opaque_len);
    }

    void event_mv_C_fixedprob_W_homo_double(hipStream_t stream, void **buffers,
                                            const char *opaque, std::size_t opaque_len) {
        event_mv_C_fixedprob_W_homo<double>(stream, buffers, opaque, opaque_len);
    }


}

