#include "hip/hip_runtime.h"
//
// Created by adadu on 2022/12/1.
//

#include "gpu_jitconn_matvec_atomic.cuh"


#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val)
        {
            unsigned long long int* address_as_ull = (unsigned long long int*)address;
            unsigned long long int old = *address_as_ull, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __longlong_as_double(old);
        }
#endif


namespace brainpy_lib {
    namespace {

        template<typename T, const int BLOCK_SIZE>
        __global__ void _jitconn_prob_homo_v2(
                const T *vector,  /* vector */
                const unsigned int conn_seed,  /* matrix */
                const float conn_prob,
                const unsigned int num_row,  /* shape */
                const unsigned int num_col,
                T *out  /* output */
        ) {
            const unsigned int col_i = blockIdx.x * blockDim.x + threadIdx.x;

            if (col_i < num_col) {

                // random state
                hiprandState state;
                hiprand_init(conn_seed + col_i, 0, 0, &state);

                // summation
                T v = vector[col_i];
                int row_i = (int) ceil(log(hiprand_uniform(&state)) / conn_prob);
                while (row_i < num_row) {
                    atomicAdd(&out[row_i], v);
                    row_i += (int) ceil(log(hiprand_uniform(&state)) / conn_prob);
                }
            }
        }


        template<typename T>
        inline void matvec_atomic_jitconn_prob_homo_v2(hipStream_t stream,
                                                       void **buffers,
                                                       const char *opaque,
                                                       std::size_t opaque_len) {
            // size
            const JITConnProbHomoDescriptor &d = *UnpackDescriptor<JITConnProbHomoDescriptor>(opaque, opaque_len);
            const unsigned int n_row = d.n_row;
            const unsigned int n_col = d.n_col;
            const unsigned int conn_seed = d.seed;
            const float conn_prob = d.prob;

            // data
            const T *vector = reinterpret_cast<const T *>(buffers[0]);
            T *y = reinterpret_cast<T *>(buffers[1]);

            // processing
            const int block_dim = 256;
            const int grid_dim = (n_col + block_dim - 1) / block_dim;
            hipMemset(y, 0, sizeof(T) * n_row);
            _jitconn_prob_homo_v2<T, block_dim><<<grid_dim, block_dim, 0, stream>>>(
                    vector, conn_seed, conn_prob, n_row, n_col, y
            );
            ThrowIfError(hipGetLastError());
        }


        template<typename T, const int BLOCK_SIZE>
        __global__ void _jitconn_prob_uniform_v2(
                const T *vector,  /* vector */
                const unsigned int conn_seed,  /* matrix */
                const float conn_prob,
                const float w_min,
                const float w_range,
                const unsigned int num_row,  /* shape */
                const unsigned int num_col,
                T *out  /* output */
        ) {
            const unsigned int col_i = blockIdx.x * blockDim.x + threadIdx.x;

            if (col_i < num_col) {

                // random state
                hiprandState state;
                hiprand_init(conn_seed + col_i, 0, 0, &state);

                // summation
                T v = vector[col_i];
                int row_i = (int) ceil(log(hiprand_uniform(&state)) / conn_prob);
                while (row_i < num_row) {
                    atomicAdd(&out[row_i], v * (hiprand_uniform(&state) * w_range + w_min));
                    row_i += (int) ceil(log(hiprand_uniform(&state)) / conn_prob);
                }
            }
        }


        template<typename T>
        inline void matvec_atomic_jitconn_prob_uniform_v2(hipStream_t stream,
                                                          void **buffers,
                                                          const char *opaque,
                                                          std::size_t opaque_len) {
            // size
            const JITConnProbUniformDescriptor &d = *UnpackDescriptor<JITConnProbUniformDescriptor>(opaque,
                                                                                                    opaque_len);
            const unsigned int n_row = d.n_row;
            const unsigned int n_col = d.n_col;
            const unsigned int conn_seed = d.seed;
            const float conn_prob = d.prob;
            const float w_min = d.w_min;
            const float w_range = d.w_range;

            // data
            const T *vector = reinterpret_cast<const T *>(buffers[0]);
            T *y = reinterpret_cast<T *>(buffers[1]);

            // processing
            const int block_dim = 256;
            const int grid_dim = (n_col + block_dim - 1) / block_dim;
            hipMemset(y, 0, sizeof(T) * n_row);
            _jitconn_prob_uniform_v2<T, block_dim><<<grid_dim, block_dim, 0, stream>>>(
                    vector, conn_seed, conn_prob, w_min, w_range, n_row, n_col, y
            );
            ThrowIfError(hipGetLastError());
        }


        template<typename T, const int BLOCK_SIZE>
        __global__ void _jitconn_prob_normal_v2(
                const T *vector,  /* vector */
                const unsigned int conn_seed,  /* matrix */
                const float log_prob,
                const float w_mu,
                const float w_sigma,
                const unsigned int num_row,  /* shape */
                const unsigned int num_col,
                T *out  /* output */
        ) {

            const unsigned int col_i = blockIdx.x * blockDim.x + threadIdx.x;

            if (col_i < num_col) {
                // random state
                hiprandState state;
                hiprand_init(conn_seed + col_i, 0, 0, &state);

                // summation
                T v = vector[col_i];
                int row_i = (int) ceil(log(hiprand_uniform(&state)) / log_prob);
                while (row_i < num_row) {
                    atomicAdd(&out[row_i], v * (hiprand_normal(&state) * w_sigma + w_mu));
                    row_i += (int) ceil(log(hiprand_uniform(&state)) / log_prob);
                }
            }
        }


        template<typename T>
        inline void matvec_atomic_jitconn_prob_normal_v2(hipStream_t stream,
                                                         void **buffers,
                                                         const char *opaque,
                                                         std::size_t opaque_len) {
            // size
            const JITConnProbNormalDescriptor &d = *UnpackDescriptor<JITConnProbNormalDescriptor>(opaque,
                                                                                                  opaque_len);
            const unsigned int n_row = d.n_row;
            const unsigned int n_col = d.n_col;
            const unsigned int conn_seed = d.seed;
            const float conn_prob = d.prob;
            const float w_mu = d.w_mu;
            const float w_sigma = d.w_sigma;

            // data
            const T *vector = reinterpret_cast<const T *>(buffers[0]);
            T *y = reinterpret_cast<T *>(buffers[1]);

            // processing
            const int block_dim = 256;
            const int grid_dim = (n_col + block_dim - 1) / block_dim;
            hipMemset(y, 0, sizeof(T) * n_row);
            _jitconn_prob_normal_v2<T, block_dim><<<grid_dim, block_dim, 0, stream>>>(
                    vector, conn_seed, conn_prob, w_mu, w_sigma, n_row, n_col, y
            );
            ThrowIfError(hipGetLastError());
        }


    }

    void matvec_atomic_jitconn_prob_homo_v2_float(hipStream_t stream, void **buffers,
                                                  const char *opaque, std::size_t opaque_len) {
        matvec_atomic_jitconn_prob_homo_v2<float>(stream, buffers, opaque, opaque_len);
    }

    void matvec_atomic_jitconn_prob_homo_v2_double(hipStream_t stream, void **buffers,
                                                   const char *opaque, std::size_t opaque_len) {
        matvec_atomic_jitconn_prob_homo_v2<double>(stream, buffers, opaque, opaque_len);
    }

    void matvec_atomic_jitconn_prob_uniform_v2_float(hipStream_t stream, void **buffers,
                                                     const char *opaque, std::size_t opaque_len) {
        matvec_atomic_jitconn_prob_uniform_v2<float>(stream, buffers, opaque, opaque_len);
    }

    void matvec_atomic_jitconn_prob_uniform_v2_double(hipStream_t stream, void **buffers,
                                                      const char *opaque, std::size_t opaque_len) {
        matvec_atomic_jitconn_prob_uniform_v2<double>(stream, buffers, opaque, opaque_len);
    }

    void matvec_atomic_jitconn_prob_normal_v2_float(hipStream_t stream, void **buffers,
                                                    const char *opaque, std::size_t opaque_len) {
        matvec_atomic_jitconn_prob_normal_v2<float>(stream, buffers, opaque, opaque_len);
    }

    void matvec_atomic_jitconn_prob_normal_v2_double(hipStream_t stream, void **buffers,
                                                     const char *opaque, std::size_t opaque_len) {
        matvec_atomic_jitconn_prob_normal_v2<double>(stream, buffers, opaque, opaque_len);
    }


}

