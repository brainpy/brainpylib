#include "hip/hip_runtime.h"
//
// Created by adadu on 2022/11/21.
//

#include "gpu_event_info.cuh"


namespace brainpy_lib {
    namespace {

        template<typename T, const int NUM_THREAD>
        __global__ void collect_spike_info(
                const std::uint32_t size,
                const T *events,
                int *event_ids,
                int *event_num
        ) {
            const int id = blockDim.x * blockIdx.x + threadIdx.x;
            const int gid = size * blockIdx.y + id;
            __shared__ unsigned int shSpk[NUM_THREAD];
            __shared__ unsigned int shPosSpk;
            __shared__ unsigned int shSpkCount;
            if (threadIdx.x == 0) {
                shSpkCount = 0;
            }
            __syncthreads();

            if (id < size) {
                if (events[gid]) {
                    const unsigned int spkIdx = atomicAdd(&shSpkCount, 1);
                    shSpk[spkIdx] = id;
                }
                __syncthreads();

                if (threadIdx.x == 0) {
                    shPosSpk = atomicAdd(&event_num[blockIdx.y], shSpkCount);
                }
                __syncthreads();

                if (threadIdx.x < shSpkCount) {
                    const int n = shSpk[threadIdx.x];
                    event_ids[blockIdx.y * size + shPosSpk + threadIdx.x] = n;
                }
            }
        }

        template<typename T>
        inline void nonzero_256(hipStream_t stream, void **buffers, const char *opaque, std::size_t opaque_len) {
            const NonZeroDescriptor &d = *UnpackDescriptor<NonZeroDescriptor>(opaque, opaque_len);
            const std::uint32_t event_size = d.event_size;
            const std::uint32_t batch_size = d.batch_size;

            const T *events = reinterpret_cast<const bool *>(buffers[0]);
            int *event_ids = reinterpret_cast<int *>(buffers[1]);
            int *event_num = reinterpret_cast<int *>(buffers[2]);

            hipMemset(event_ids, -1, sizeof(int) * event_size * batch_size);
            hipMemset(event_num, 0, sizeof(int) * batch_size);
            dim3 grid((event_size + 255) / 256, batch_size);
            collect_spike_info<256><<<grid, 256, 0, stream>>>(event_size, events, event_ids, event_num);
            ThrowIfError(hipGetLastError());
        }


    }


    void nonzero_bool(hipStream_t stream, void **buffers, const char *opaque, std::size_t opaque_len) {
        nonzero_256<bool>(stream, buffers, opaque, opaque_len);
    }


    void nonzero_int(hipStream_t stream, void **buffers, const char *opaque, std::size_t opaque_len) {
        nonzero_256<int>(stream, buffers, opaque, opaque_len);
    }


    void nonzero_long(hipStream_t stream, void **buffers, const char *opaque, std::size_t opaque_len) {
        nonzero_256<long>(stream, buffers, opaque, opaque_len);
    }


    void nonzero_float(hipStream_t stream, void **buffers, const char *opaque, std::size_t opaque_len) {
        nonzero_256<float>(stream, buffers, opaque, opaque_len);
    }

    void nonzero_double(hipStream_t stream, void **buffers, const char *opaque, std::size_t opaque_len) {
        nonzero_256<double>(stream, buffers, opaque, opaque_len);
    }


}


